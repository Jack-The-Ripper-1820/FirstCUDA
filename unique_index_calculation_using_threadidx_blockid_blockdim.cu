#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void unique_idx_calc_threadIdx(int* input) {
	int tid = threadIdx.x;
	printf("threadIdx : %d, value : %d \n", tid, input[tid]);
}

__global__ void unique_gid_calculation(int* input) {
	int tid = threadIdx.x;
	int offset = blockIdx.x * blockDim.x;
	int gid = tid + offset;
	printf("blockIdx.x : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, threadIdx.x, gid, input[gid]);
}

__global__ void unique_gid_calculation_2d(int* input) {
	int tid = threadIdx.x;
	int blockoffset = blockIdx.x * blockDim.x + blockIdx.y * blockDim.y;
	int rowoffset = blockIdx.y * blockDim.x * gridDim.x;
	int gid = tid + blockoffset + rowoffset;

	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, blockIdx.y, threadIdx.x, gid, input[gid]);
}

__global__ void unique_gid_calculation_2d_2d(int* input) {
	int tid = threadIdx.x + blockDim.x * threadIdx.y;
	int blockoffset = blockIdx.x * blockDim.x * blockDim.y;
	int rowoffset = blockIdx.y * blockDim.x * blockDim.y * gridDim.x;
	int gid = tid + blockoffset + rowoffset;

	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, gid : %d, value : %d \n",
		blockIdx.x, blockIdx.y, threadIdx.x, gid, input[gid]);
}

//int main() {
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 82, 281, 2, 1, 93, 5, 91, 57, 542, 134, 633, 76, 73, 134, 63, 313};
//
//	for (int i = 0; i < array_size; i++) {
//		printf("%d ", h_data[i]);
//	}
//	printf("\n \n");
//
//	int* d_data;
//	cudaMalloc((void**)&d_data, array_byte_size);
//	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(2, 2);
//	dim3 grid(2, 2);
//
//	//unique_idx_calc_threadIdx << <grid, block >> > (d_data);
//	//unique_gid_calculation << <grid, block >> > (d_data);
//	//unique_gid_calculation_2d << <grid, block >> > (d_data);
//	unique_gid_calculation_2d_2d << <grid, block >> > (d_data);
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	
//	return 0;
//}