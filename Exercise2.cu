#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

__global__ void unique_gid_calculation_2d_2d_2d(int* input, int size) {
	int blockId = blockIdx.x + blockIdx.y * gridDim.x 
		+ gridDim.x * gridDim.y * blockIdx.z;
	int gid = blockId * (blockDim.x * blockDim.y * blockDim.z)
		+ (threadIdx.z * (blockDim.x * blockDim.y))
		+ (threadIdx.y * blockDim.x) + threadIdx.x;
	
	printf("blockIdx.x : %d, blockIdx.y : %d, threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d, gid : %d, value : %d \n",
		blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y, threadIdx.z, gid, input[gid]);
}


//int main() {
//	int size = 512;
//	int byte_size = size * sizeof(int);
//
//	int* h_input;
//	h_input = (int*)malloc(byte_size);
//
//	time_t t;
//	srand((unsigned)time(&t));
//
//	for (int i = 0; i < size; i++) {
//		h_input[i] = (int)(rand() & 0xff);
//	}
//
//	int* d_input;
//	cudaMalloc((void**)&d_input, byte_size);
//
//	cudaMemcpy(d_input, h_input, byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(2,2,2);
//	dim3 grid(4,4,4);
//
//	unique_gid_calculation_2d_2d_2d << <grid, block >> > (d_input, size);
//	cudaDeviceSynchronize();
//
//	cudaFree(d_input);
//	free(h_input);
//
//	cudaDeviceReset();
//	return 0;
//}